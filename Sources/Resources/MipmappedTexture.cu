#include "hip/hip_runtime.h"
#include "Resources/MipmappedTexture.cuh"

using namespace Maths;
using namespace Resources;

__host__ __device__ IVec2 MipmappedTexture::GetResolution() const
{
	return resolution;
}

__device__ Vec4 MipmappedTexture::Sample(Vec2 uv) const
{
	float4 res = tex2D<float4>(device_tex, uv.x, uv.y);
	return Vec4(res.x, res.y, res.z, res.w);
}

__device__ Vec4 MipmappedTexture::Sample(Vec2 uv, Vec2 dX, Vec2 dY) const
{
	float2 a = {dX.x, dX.y};
	float2 b = {dY.x, dY.y};
	float4 res = tex2DGrad<float4>(device_tex, uv.x, uv.y, a, b);
	return Vec4(res.x, res.y, res.z, res.w);
}
