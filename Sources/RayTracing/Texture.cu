#include "hip/hip_runtime.h"
#include "RayTracing/Texture.cuh"

using namespace Maths;
using namespace RayTracing;

__host__ __device__ IVec2 Texture::GetResolution() const
{
	return resolution;
}

__device__ Vec4 Texture::Sample(Vec2 uv)
{
	float4 res = tex2D<float4>(device_tex, uv.x, uv.y);
	for (u8 i = 0; i < 4; ++i)
	{
		*(&res.x + i) = powf(*(&res.x + i), 1/2.3f);
	}
	return Vec4(res.x, res.y, res.z, res.w);
}
